#include "hip/hip_runtime.h"
#include "spatial.h"
#include "simple_knn.h"

torch::Tensor
distCUDA2(const torch::Tensor& points)
{
  const int P = points.size(0);

  auto float_opts = points.options().dtype(torch::kFloat32);
  torch::Tensor means = torch::full({P}, 0.0, float_opts);
  
  SimpleKNN::knn(P, (float3*)points.contiguous().data<float>(), means.contiguous().data<float>());

  return means;
}